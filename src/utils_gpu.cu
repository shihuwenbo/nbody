#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// allocate memory on gpu
extern "C++" void cu_safe_falloc(float **g_f, size_t n_elem) {
    void *gptr;
    hipError_t crc = hipMalloc(&gptr, n_elem*sizeof(float));
    if(crc) {
        printf("hipMalloc Error=%d:%s\n", crc, hipGetErrorString(crc));
        exit(1);
    }
    *g_f = (float*) gptr;
}

// free memory on gpu
extern "C++" void cu_free(void *g_d) {
   hipError_t crc = hipFree(g_d);
   if (crc) {
      printf("hipFree Error=%d:%s\n", crc, hipGetErrorString(crc));
      exit(1);
   }
}

// copy from cpu space f to gpu space g_f
extern "C++" void memcpy_htod(float *g_f, float *f, size_t n_elem) {
   hipError_t crc = hipMemcpy((void*)g_f, f, sizeof(float)*n_elem,
                    hipMemcpyHostToDevice);
   if (crc) {
      printf("hipMemcpyHostToDevice float Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
}

// copy from gpu space g_f to cpu space f
extern "C++" void memcpy_dtoh(float *f, float *g_f, size_t n_elem) {
   hipError_t crc = hipMemcpy(f, (void*)g_f, sizeof(float)*n_elem,
                    hipMemcpyDeviceToHost);
   if (crc) {
      printf("hipMemcpyDeviceToHost float Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}
